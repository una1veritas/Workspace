#include "hip/hip_runtime.h"
/*
============================================================================
Name        : fft_main.c
Author      : Sin Shimozono
Version     :
Copyright   : reserved.
Description : Factored discrete Fourier transform, or FFT, and its inverse iFFT
============================================================================
* Reference:
* http://www.math.wustl.edu/~victor/mfmm/fourier/fft.c
* http://rosettacode.org/wiki/Fast_Fourier_transform
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>


#define TEXTBUFFER_LENGTH 512
#define VECTOR_MAXSIZE 512

struct dcompvec {
	hipfftComplex * elem;
	int dimsize;
};
typedef struct dcompvec dcompvec;
const int CONJ_REVERSE = 1;

struct text {
	char str[TEXTBUFFER_LENGTH];
	int length;
};
typedef struct text text;

__global__ void cuCvecmul_into(hipComplex * x, hipComplex * y, const int dimsize);
__global__ void cuCfindpos(hipComplex * v, const int dimsize, const int pattlen);
long smallestpow2(const long n) {
	long t = 1;
	while (t < n) {
		t <<= 1;
	}
	return t;
}

#define min(x,y)  ( ((x) < (y) ? (x) : (y)) )
#define max(x,y)  ( ((x) < (y) ? (y) : (x)) )
#define abs(x)  ( (x) < 0 ? (-(x)) : (x) )

int make_signal(text * text1, int dimsize, dcompvec * vec, const int flag);
void print_vector(const char *title, hipfftComplex *x, int n);

void cufft(hipfftComplex * vec, int n, int inverseflag);

__host__ __device__ static __inline__ hipComplex cuCexpf(hipComplex x)
{
	float factor = expf(x.x);
	return make_hipComplex(factor * cosf(x.y), factor * sinf(x.y));
}

int main(int argc, char * argv[]) {
	text text1, text2;
	dcompvec vec1, vec2;
	int vecsize;
	int pattlen;

	if ( argc != 3)
		exit(EXIT_FAILURE);

	strncpy(text1.str, argv[1], TEXTBUFFER_LENGTH);
	text1.length = min(strlen(text1.str), TEXTBUFFER_LENGTH);
	strncpy(text2.str, argv[2], TEXTBUFFER_LENGTH);
	text2.length = min(strlen(text2.str), TEXTBUFFER_LENGTH);

	printf("inputs: \"%s\", \"%s\" \n", text1.str, text2.str);
	vecsize = smallestpow2(min(VECTOR_MAXSIZE, max(text1.length, text2.length)));
	pattlen = min(text1.length, text2.length);

	make_signal(&text1, vecsize, &vec1, !CONJ_REVERSE);
	make_signal(&text2, vecsize, &vec2, CONJ_REVERSE);
	/* FFT, iFFT of v[]: */
	print_vector("text1 ", vec1.elem, vec1.dimsize);
	print_vector("text2 ", vec2.elem, vec2.dimsize);

	/*�^�C�}�[���쐬���Čv���J�n*/
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	/* GPU�p���������蓖�� */
	hipfftComplex *devmemptr;
	/* �o�b�`�� 2 (vec1.elem, vec2.elem)*/
	hipMalloc((void**)&devmemptr, sizeof(hipfftComplex) * vecsize * 2 );

	/* GPU�p�������ɓ]�� */
	hipMemcpy(devmemptr, vec1.elem, sizeof(hipfftComplex)* vecsize, hipMemcpyHostToDevice);
	hipMemcpy(devmemptr+vecsize, vec2.elem, sizeof(hipfftComplex)* vecsize, hipMemcpyHostToDevice);

	/* 1D FFT plan�쐬 */
	hipfftHandle plan2way, plan1inv;
	hipfftPlan1d(&plan2way, vecsize, HIPFFT_C2C, 2);
	hipfftPlan1d(&plan1inv, vecsize, HIPFFT_C2C, 1);

	hipfftExecC2C(plan2way, devmemptr, devmemptr, HIPFFT_FORWARD);

#ifdef DEBUG
	/* �v�Z���ʂ�GPU����������]�����ĕ\�� */
	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)*vec1.dimsize, hipMemcpyDeviceToHost);
	hipMemcpy(vec2.elem, devmemptr+vecsize, sizeof(hipfftComplex)*vec2.dimsize, hipMemcpyDeviceToHost);
	print_vector("fft1: 2 ", vec1.elem, vec1.dimsize);
	print_vector("fft2 ", vec2.elem, vec2.dimsize);
#endif

	/* �x�N�g���̐ς��Ƃ� */
	dim3 grid(16, 1);
	dim3 block(VECTOR_MAXSIZE/16, 1);
	cuCvecmul_into <<< grid, block >>> (devmemptr, devmemptr+vecsize, vecsize);

#ifdef DEBUG
	/* �v�Z���ʂ�GPU����������]�����ĕ\�� */
	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)* vecsize, hipMemcpyDeviceToHost);
	hipMemcpy(vec2.elem, devmemptr + vecsize, sizeof(hipfftComplex)*vecsize, hipMemcpyDeviceToHost);
	print_vector("prod ", vec1.elem, vec1.dimsize);
#endif

	hipfftExecC2C(plan1inv, devmemptr, devmemptr, HIPFFT_BACKWARD);


	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)* vecsize, hipMemcpyDeviceToHost);
	print_vector("iFFT ", vec1.elem, vecsize);

	cuCfindpos<<<grid, block>>>(devmemptr, vecsize, pattlen);

	/*
	int pos = vec1.dimsize;
	for (int i = pattlen - 1; i < vec1.dimsize + pattlen - 1; i++) {
		if (abs(hipCrealf(vec1.elem[(vec1.dimsize + i - 1) % vec1.dimsize])/vecsize - (double)pattlen) < (1.0F/4096.0F))
			pos = min(pos, i);
	}
	*/
	/*
	printf("Occurring positions: ");
	for(int i = 0; i < vec1.dimsize; i++) {
	if ( abs(creal(vec1.elem[(vec1.dimsize - pattlen + i) % vec1.dimsize]) - (double) pattlen) < 0.0001220703125 )
	printf("%d, ", i - pattlen + 1);
	}
	printf(".\n");
	*/
	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)* vecsize, hipMemcpyDeviceToHost);
	int pos = (int) hipCrealf(vec1.elem[0]);

	/*�^�C�}�[���~�������������Ԃ�\��*/
	sdkStopTimer(&timer);
	printf("�v�Z���� =%f(ms)\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	print_vector(" pos ", vec1.elem, vecsize);
	if (pos < vecsize) {
		printf("The 1st occurrence at: %d.\n", pos);
	}
	else {
		printf("None.\n");
	}

	/* GPU�p�������J�� */
	hipFree(devmemptr);

	/* CUFFT plan�폜 */
	hipfftDestroy(plan2way);
	hipfftDestroy(plan2way);

	free(vec1.elem);
	free(vec2.elem);

	exit(EXIT_SUCCESS);
}


int make_signal(text * str, const int dimsize, dcompvec * vec, const int flag) {
	int len;
	int dst;
	float factor;

	// the first as normal
	vec->dimsize = dimsize;
	vec->elem = (hipComplex*)malloc(sizeof(hipComplex)*dimsize);
	len = str->length;
	for (int i = 0; i < vec->dimsize; ++i) {
		if (!flag) {
			dst = i;
			factor = 2 * 3.14159265358979323846264338327950288F;
		}
		else {
			dst = vec->dimsize - i - 1;
			factor = -2 * 3.14159265358979323846264338327950288F;
		}
		if (i < len)
			vec->elem[dst] = cuCexpf(make_hipComplex(0, factor * (float)(str->str[i]) / 256.0f));  // by rotated unit vector
																								  // (*array)[i] = (float)(str[i]) / 128.0f  ;  // by char value
		else
			vec->elem[dst] = make_hipComplex(0, 0);
	}
	return 1;
}

/* Print a vector of complexes as ordered pairs. */
void print_vector(const char *title, hipfftComplex *x, int n) {
	int i;
	printf("%s (dim=%d):\n", title, n);
	for (i = 0; i < n; i++)
		printf("%5d    ", i);
	putchar('\n');
	for (i = 0; i < n; i++)
		printf(" %7.3f,", hipCrealf(x[i]));
	putchar('\n');
	for (i = 0; i < n; i++)
		printf(" %7.3f,", hipCimagf(x[i]));
	putchar('\n');
	for (i = 0; i < n; i++)
		printf(" %7.3f,", hipCabsf(x[i]));
	printf("\n\n");
	return;
}

__global__ void cuCvecmul_into(hipComplex * v, hipComplex * w, const int dimsize) {
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	if ( idx < dimsize )
		v[idx] = hipCmulf(v[idx], w[idx]);
	__syncthreads();
}

__global__ void cuCfindpos(hipComplex * v, const int dimsize, const int pattlen) {
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	float val;
	int width;

	//(vec1.dimsize - pattlen + i) % vec1.dimsize

	if (idx < dimsize) {
		int i = dimsize - pattlen + idx;
		if (i >= dimsize)
			i %= dimsize;
		val = hipCrealf(v[i]) / dimsize - (float)pattlen;
		if (val < 0)
			val = -val;
	}
	__syncthreads();
	if (idx < dimsize) {
		if (val < 0.000122)
			v[idx] = make_hipComplex(idx, 0.0);
		else
			v[idx] = make_hipComplex(dimsize, 0.0);
	}
	__syncthreads();

	float pos1, pos2;
	for (width = (dimsize >> 1); width > 0; width >>= 1) {
		if ( idx < width ) {
			pos1 = hipCrealf(v[idx]);
			pos2 = hipCrealf(v[idx + width]);
			v[idx] = make_hipComplex( (pos1 < pos2 ? pos1 : pos2), 0.0);
		}
		__syncthreads();
	}
	
}

/* 2567
So Satan spake, and him Beelzebub
Thus answer'd. Leader of those Armies bright,
Which but th' Onmipotent none could have foyld,
If once they hear that voyce, thir liveliest pledge
Of hope in fears and dangers, heard so oft 
In worst extreams, and on the perilous edge
Of battel when it rag'd, in all assaults
Thir surest signal, they will soon resume
New courage and revive, though now they lye
Groveling and prostrate on yon Lake of Fire, 
As we erewhile, astounded and amaz'd,
No wonder, fall'n such a pernicious highth.
He scarce had ceas't when the superiour Fiend
Was moving toward the shoar; his ponderous shield
Ethereal temper, massy, large and round, 
Behind him cast; the broad circumference
Hung on his shoulders like the Moon, whose Orb
Through Optic Glass the Tuscan Artist views
At Ev'ning from the top of Fesole,
Or in Valdarno, to descry new Lands, 
Rivers or Mountains in her spotty Globe.
His Spear, to equal which the tallest Pine
Hewn on Norwegian hills, to be the Mast
Of some great Ammiral, were but a wand,
He walkt with to support uneasie steps 
Over the burning Marle, not like those steps
On Heavens Azure, and the torrid Clime
Smote on him sore besides, vaulted with Fire;
Nathless he so endur'd, till on the Beach
Of that inflamed Sea, he stood and call'd 
His Legions, Angel Forms, who lay intrans't
Thick as Autumnal Leaves that strow the Brooks
In Vallombrosa, where th' Etrurian shades
High overarch't imbowr; or scatterd sedge
Afloat, when with fierce Winds Orion arm'd 
Hath vext the Red-Sea Coast, whose waves orethrew
Busiris and his Memphian Chivalry,
While with perfidious hatred they pursu'd
The Sojourners of Goshen, who beheld
From the safe shore thir floating Carkases 
And broken Chariot Wheels, so thick bestrown
Abject and lost lay these, covering the Flood,
Under amazement of thir hideous change.
He call'd so loud, that all the hollow Deep
Of Hell resounded. Princes, Potentates, 
Warriers, the Flowr of Heav'n, once yours, now lost,
If such astonishment as this can sieze
Eternal spirits; or have ye chos'n this place
After the toyl of Battel to repose
Your wearied vertue, for the ease you find 
To slumber here, as in the Vales of Heav'n?
Or in this abject posture have ye sworn
To adore the Conquerour? who now beholds
Cherube and Seraph rowling in the Flood
With scatter'd Arms and Ensigns, till anon 
His swift pursuers from Heav'n Gates discern
Th' advantage, and descending tread us down
Thus drooping, or with linked Thunderbolts
Transfix us to the bottom of this Gulfe.
Awake, arise, or be for ever fall'n. 
*/