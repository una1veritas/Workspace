#include "hip/hip_runtime.h"
/*
============================================================================
Name        : fft_main.c
Author      : Sin Shimozono
Version     :
Copyright   : reserved.
Description : Factored discrete Fourier transform, or FFT, and its inverse iFFT
============================================================================
* Reference:
* http://www.math.wustl.edu/~victor/mfmm/fourier/fft.c
* http://rosettacode.org/wiki/Fast_Fourier_transform
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>


#define VECTOR_MAXSIZE 512

struct compvect {
	hipfftComplex * elem;
	unsigned int dim;
};
typedef struct compvect compvect;
const int CONJ_REVERSE = 1;

struct text {
	char * str;
	unsigned int size;
	unsigned int length;
};
typedef struct text text;

__global__ void cuCvecmul_into(hipComplex * x, hipComplex * y, const int dimsize);
__global__ void cuCfindpos(hipComplex * v, int * occurrences, const int dim, const int pattlen);
long smallestpow2(const long n) {
	long t = 1;
	while (t < n) {
		t <<= 1;
	}
	return t;
}

#define min(x,y)  ( ((x) < (y) ? (x) : (y)) )
#define max(x,y)  ( ((x) < (y) ? (y) : (x)) )
#define abs(x)  ( (x) < 0 ? (-(x)) : (x) )

int make_signal(const text * text1, const int dimsize, compvect * vec, const int flag);
void print_vector(const char *title, compvect *x);

__host__ __device__ static __inline__ hipComplex cuCexpf(hipComplex x)
{
	float factor = expf(x.x);
	return make_hipComplex(factor * cosf(x.y), factor * sinf(x.y));
}

int main(int argc, char * argv[]) {
	text text1, text2;
	compvect vec1, vec2;
	int vecsize;
	int pattlen;

	if (argc != 3)
		exit(EXIT_FAILURE);

	text1.size = VECTOR_MAXSIZE;
	text1.str = (char*)malloc(sizeof(char)*text1.size);
	strncpy(text1.str, argv[1], text1.size);
	text1.length = min(strlen(argv[1]), text1.size);
	text2.size = VECTOR_MAXSIZE;
	text2.str = (char*)malloc(sizeof(char)*text2.size);
	strncpy(text2.str, argv[2], text2.size);
	text2.length = min(strlen(argv[2]), text2.size);

	printf("inputs: \"%s\", \"%s\" \n", text1.str, text2.str);
	vecsize = smallestpow2(min(max(text1.length, text2.length), VECTOR_MAXSIZE));
	pattlen = min(text1.length, text2.length);

	make_signal(&text1, vecsize, &vec1, !CONJ_REVERSE);
	make_signal(&text2, vecsize, &vec2, CONJ_REVERSE);
	/* FFT, iFFT of v[]: */
	print_vector("text1 ", &vec1);
	print_vector("text2 ", &vec2);

	/* ���Ԍv���p�^�C�}�[�̃Z�b�g�A�b�v�ƌv���J�n */
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	/* GPU�p���������蓖�� */
	hipfftComplex *devmemptr;
	//	int * devresptr;
	/* �o�b�`�� 2 (vec1.elem, vec2.elem)�@*/
	hipMalloc((void**)&devmemptr, sizeof(hipfftComplex) * vecsize * 2);
	//	hipMalloc((void**)&devresptr, sizeof(int) * vecsize);

	/* GPU�p�������ɓ]�� */
	hipMemcpy(devmemptr, vec1.elem, sizeof(hipfftComplex)* vecsize, hipMemcpyHostToDevice);
	hipMemcpy(devmemptr + vecsize, vec2.elem, sizeof(hipfftComplex)* vecsize, hipMemcpyHostToDevice);

	/* 1D FFT plan�쐬 */
	hipfftHandle plan2way, plan1inv;
	hipfftPlan1d(&plan2way, vecsize, HIPFFT_C2C, 2);
	hipfftPlan1d(&plan1inv, vecsize, HIPFFT_C2C, 1);

	hipfftExecC2C(plan2way, devmemptr, devmemptr, HIPFFT_FORWARD);

#ifdef DEBUG
	/*  �v�Z���ʂ�GPU����������]�����ĕ\�� */
	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)*vec1.dimsize, hipMemcpyDeviceToHost);
	hipMemcpy(vec2.elem, devmemptr + vecsize, sizeof(hipfftComplex)*vec2.dimsize, hipMemcpyDeviceToHost);
	print_vector("fft1: 2 ", vec1.elem, vec1.dimsize);
	print_vector("fft2 ", vec2.elem, vec2.dimsize);
#endif

	/* �x�N�g���̐ς��Ƃ� */
	dim3 grid(16, 1);
	dim3 block(VECTOR_MAXSIZE / 16, 1);
	cuCvecmul_into << < grid, block >> > (devmemptr, devmemptr + vecsize, vecsize);

#ifdef DEBUG
	/*  �v�Z���ʂ�GPU����������]�����ĕ\�� */
	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)* vecsize, hipMemcpyDeviceToHost);
	hipMemcpy(vec2.elem, devmemptr + vecsize, sizeof(hipfftComplex)*vecsize, hipMemcpyDeviceToHost);
	print_vector("prod ", vec1.elem, vec1.dimsize);
#endif

	hipfftExecC2C(plan1inv, devmemptr, devmemptr, HIPFFT_BACKWARD);


	hipMemcpy(vec1.elem, devmemptr, sizeof(hipfftComplex)* vecsize, hipMemcpyDeviceToHost);
	print_vector("iFFT ", &vec1);

	cuCfindpos << <grid, block >> >(devmemptr, (int *)(devmemptr + vecsize), vecsize, pattlen);

	/*
	int pos = vec1.dimsize;
	for (int i = pattlen - 1; i < vec1.dimsize + pattlen - 1; i++) {
	if (abs(hipCrealf(vec1.elem[(vec1.dimsize + i - 1) % vec1.dimsize])/vecsize - (double)pattlen) < (1.0F/4096.0F))
	pos = min(pos, i);
	}
	*/
	/*
	printf("Occurring positions: ");
	for(int i = 0; i < vec1.dimsize; i++) {
	if ( abs(creal(vec1.elem[(vec1.dimsize - pattlen + i) % vec1.dimsize]) - (double) pattlen) < 0.0001220703125 )
	printf("%d, ", i - pattlen + 1);
	}
	printf(".\n");
	*/
	int * pos = (int *)malloc(sizeof(int)*vecsize);
	hipMemcpy(pos, devmemptr + vecsize, sizeof(int) * vecsize, hipMemcpyDeviceToHost);

	/* �^�C�}�[���~�������������Ԃ�\�� */
	sdkStopTimer(&timer);
	printf("computation time %f (ms)\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	printf("\nResult: \n");
	for (int i = 0; i < vecsize; i++) {
		printf("[%d] = %d, ", i, pos[i]);
	}
	printf("\n");
	if (pos[0] < vecsize)
		printf("First occurrence is at %d.\n", pos[0] - pattlen + 1);
	else
		printf("Could not find.\n");

	/* GPU�p�������J��*/
	hipFree(devmemptr);
	//	hipFree(devresptr);

	/* CUFFT plan�폜 */
	hipfftDestroy(plan2way);
	hipfftDestroy(plan2way);

	free(pos);
	free(vec1.elem);
	free(vec2.elem);
	free(text1.str);
	free(text2.str);

	exit(EXIT_SUCCESS);
}


int make_signal(const text * str, const int dim, compvect * vec, const int flag) {
	int dst;
	float factor;

	// the first as normal
	vec->elem = (hipComplex*)malloc(sizeof(hipComplex)*dim);
	vec->dim = dim;
	for (int i = 0; i < vec->dim; ++i) {
		if (!flag) {
			dst = i;
			factor = 2 * 3.14159265358979323846264338327950288F;
		}
		else {
			dst = vec->dim - i - 1;
			factor = -2 * 3.14159265358979323846264338327950288F;
		}
		if (i < str->length)
			vec->elem[dst] = cuCexpf(make_hipComplex(0, factor * (float)(str->str[i]) / 256.0f));  // by rotated unit vector
																								  // (*array)[i] = (float)(str[i]) / 128.0f  ;  // by char value
		else
			vec->elem[dst] = make_hipComplex(0, 0);
	}
	return 1;
}

/* Print a vector of complexes as ordered pairs. */
void print_vector(const char *title, compvect *v) {
	unsigned int i;
	printf("%s (dim=%d):\n", title, v->dim);
	for (i = 0; i < v->dim; i++)
		printf("%5d    ", i);
	putchar('\n');
	for (i = 0; i < v->dim; i++)
		printf(" %7.3f,", hipCrealf(v->elem[i]));
	putchar('\n');
	for (i = 0; i < v->dim; i++)
		printf(" %7.3f,", hipCimagf(v->elem[i]));
	putchar('\n');
	for (i = 0; i < v->dim; i++)
		printf(" %7.3f,", hipCabsf(v->elem[i]));
	printf("\n\n");
	return;
}

__global__ void cuCvecmul_into(hipComplex * v, hipComplex * w, const int dimsize) {
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	if (idx < dimsize)
		v[idx] = hipCmulf(v[idx], w[idx]);
	__syncthreads();
}

__global__ void cuCfindpos(hipComplex * v, int * occ, const int dim, const int pattlen) {
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	float val;
	int width;

	//(vec1.dimsize - pattlen + i) % vec1.dimsize

	if (idx < dim) {
		val = (hipCrealf(v[idx]) / dim) - (float)pattlen;
		if (val < 0)
			val = -val;
	}
	__syncthreads();

	if (idx < dim) {
		if (val < 0.000122)
			occ[idx] = (idx + pattlen) % dim;
		else
			occ[idx] = dim;
	}
	__syncthreads();


	int t1, t2;
	for (width = (dim >> 1); width > 0; width >>= 1) {
		if (idx < width) {
			t1 = occ[idx];
			t2 = occ[idx + width];
			if (t1 < t2) {
				occ[idx] = t1;
				occ[idx + width] = t1;
			}
			else {
				occ[idx] = t2;
				occ[idx + width] = t2;
			}
		}
		__syncthreads();
	}

}

/* 2567
So Satan spake, and him Beelzebub
Thus answer'd. Leader of those Armies bright,
Which but th' Onmipotent none could have foyld,
If once they hear that voyce, thir liveliest pledge
Of hope in fears and dangers, heard so oft
In worst extreams, and on the perilous edge
Of battel when it rag'd, in all assaults
Thir surest signal, they will soon resume
New courage and revive, though now they lye
Groveling and prostrate on yon Lake of Fire,
As we erewhile, astounded and amaz'd,
No wonder, fall'n such a pernicious highth.
He scarce had ceas't when the superiour Fiend
Was moving toward the shoar; his ponderous shield
Ethereal temper, massy, large and round,
Behind him cast; the broad circumference
Hung on his shoulders like the Moon, whose Orb
Through Optic Glass the Tuscan Artist views
At Ev'ning from the top of Fesole,
Or in Valdarno, to descry new Lands,
Rivers or Mountains in her spotty Globe.
His Spear, to equal which the tallest Pine
Hewn on Norwegian hills, to be the Mast
Of some great Ammiral, were but a wand,
He walkt with to support uneasie steps
Over the burning Marle, not like those steps
On Heavens Azure, and the torrid Clime
Smote on him sore besides, vaulted with Fire;
Nathless he so endur'd, till on the Beach
Of that inflamed Sea, he stood and call'd
His Legions, Angel Forms, who lay intrans't
Thick as Autumnal Leaves that strow the Brooks
In Vallombrosa, where th' Etrurian shades
High overarch't imbowr; or scatterd sedge
Afloat, when with fierce Winds Orion arm'd
Hath vext the Red-Sea Coast, whose waves orethrew
Busiris and his Memphian Chivalry,
While with perfidious hatred they pursu'd
The Sojourners of Goshen, who beheld
From the safe shore thir floating Carkases
And broken Chariot Wheels, so thick bestrown
Abject and lost lay these, covering the Flood,
Under amazement of thir hideous change.
He call'd so loud, that all the hollow Deep
Of Hell resounded. Princes, Potentates,
Warriers, the Flowr of Heav'n, once yours, now lost,
If such astonishment as this can sieze
Eternal spirits; or have ye chos'n this place
After the toyl of Battel to repose
Your wearied vertue, for the ease you find
To slumber here, as in the Vales of Heav'n?
Or in this abject posture have ye sworn
To adore the Conquerour? who now beholds
Cherube and Seraph rowling in the Flood
With scatter'd Arms and Ensigns, till anon
His swift pursuers from Heav'n Gates discern
Th' advantage, and descending tread us down
Thus drooping, or with linked Thunderbolts
Transfix us to the bottom of this Gulfe.
Awake, arise, or be for ever fall'n.
*/