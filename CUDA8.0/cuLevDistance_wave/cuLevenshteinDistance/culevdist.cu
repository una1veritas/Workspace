#include "hip/hip_runtime.h"

/*
 * culevdist.c
 *
 *  Created on: 2016/11/26
 *      Author: sin
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "culevdist.h"

#include "debug_table.h"

#define MAX_THREADSPERBLOCK 1024

#define min(x, y)  ((x) <= (y)? (x) : (y))
#define max(x, y)  ((x) >= (y)? (x) : (y))
#define align(base, val)    ((((val)+(base)-1)/(base))*(base))

static char grayscale[] = "@#$B%8&WM*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\|()1{}[]?-_+~<>ilI;!:,\"^`'. ";

int cuStatCheck(const hipError_t stat, const char * msg) {
	if (stat != hipSuccess) {
		fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(stat));
		fflush(stderr);
		return -1;
	}
	return 0;
}


long cu_lvdist(long * inbound, long * outbound, const char t[], const long n, const char p[], const long m) {
	long result = n + m + 1; // an impossible value

	hipError_t cuStat;

	char * devt, *devp;
	long * devwavebuff, *devframe;

	cuStat = hipMalloc((void**) &devt, n);
	hipMemcpy(devt, t, n, hipMemcpyHostToDevice);
	cuStatCheck(cuStat, "hipMalloc devt");
	cuStat = hipMalloc((void**) &devp, m);
	hipMemcpy(devp, p, m, hipMemcpyHostToDevice);
	cuStatCheck(cuStat, "hipMalloc devp");


	long * devweftbuff, * devinframe, *devoutframe;
	const long table_height = m + 1;
	const long table_width = n + m + 1;

	cuStat = hipMalloc((void**)&devweftbuff, sizeof(long)*table_height*4);
	cuStatCheck(cuStat, "hipMalloc devtable failed.\n");

	long * dptable;
	long * devtable = NULL;
#ifdef DEBUG_TABLE
	cuStat = hipMalloc((void**)&devtable, sizeof(long)*table_height*table_width);
	cuStatCheck(cuStat, "hipMalloc devtable failed.\n");
#endif
	cuStat = hipMalloc((void**)&devinframe, sizeof(long)*(n + m + 1));
	cuStatCheck(cuStat, "hipMalloc devinframe failed.\n");
	cuStat = hipMalloc((void**)&devoutframe, sizeof(long)*(n + 1 + m));
	cuStatCheck(cuStat, "hipMalloc devoutframe failed.\n");
	hipMemcpy(devinframe, inbound, sizeof(long)*(n + m + 1), hipMemcpyHostToDevice);

	fprintf(stdout, "copied input, calling kernel...\n");
	fflush(stdout);

	long nthreads = align(32, m+1);
	dim3 grids(max(1, nthreads / MAX_THREADSPERBLOCK), 1), blocks(MAX_THREADSPERBLOCK);
	fprintf(stdout,"num threads %d, %d blocks.\n",nthreads, max(1, nthreads / MAX_THREADSPERBLOCK));

	cu_dptable<<< grids, blocks >>>(devweftbuff, devinframe, devoutframe, devt, n, devp, m, devtable);

	// Check for any errors launching the kernel
	cuStat = hipGetLastError();
	if (cuStat != hipSuccess) {
		fprintf(stderr, "kernel function(s) failed: %s\n", hipGetErrorString(cuStat));
		fflush(stdout);
	}
	fprintf(stdout,"Finished kernel functions.\n");
	fflush(stdout);

	hipMemcpy(outbound, devoutframe, sizeof(long)*(n + m + 1), hipMemcpyDeviceToHost);

#ifdef DEBUG_TABLE
	long * table;
	table = (long*)malloc(sizeof(long)*table_height*table_width);
	hipMemcpy(table, devtable, sizeof(long)*table_height*table_width, hipMemcpyDeviceToHost);
	// show DP table
	long c, r, dix;
	const long scales = strlen(grayscale) - 1;
	for (r = 0; r < m + 1; r++) {
		for (c = 0; c < n + 1; c++) {
			long gray = m - table[c*(m+1)+r];
			gray = (gray > 0 ? gray : 0);
			gray = (gray < 0 ? 0 : gray)*scales / m;
			fprintf(stdout, "%3ld ", table[c*(m+1)+r]);
			//fprintf(stdout, "%c ", grayscale[gray]);
		}
		fprintf(stdout, "\n");
	}
	fprintf(stdout, "\n");
	free(table);
	hipFree(devtable);
#endif

	hipFree(devweftbuff);
	hipFree(devinframe);
	hipFree(devoutframe);

	result = outbound[n];

	return result;
}

__global__ void cu_init_row(long * row, const long n, const long m) {
	long thix = blockDim.x * blockIdx.x + threadIdx.x;

	for (int rep = 0; rep < n / m; ++rep) {
		if (rep*(m + 1) + thix < n + 1) {
			row[thix] = thix;
		}
	}
	if (thix < m) {
		row[n+2+thix] = thix+1;
	}
	__syncthreads();
}

// assuming the table array size (n+1) x (m+1)
__global__ void cu_dptable(long * weftbuff, const long * inframe, long * outframe, 
	const char t[], const long n, const char p[], const long m, long * table
) {
	long dcol; // , drow; // diagonal column index
	long col; // inner diagonal index
	long ins, del, repl, cellval; // nextrepl, above, prevval;

	long *w0, *w1, *w2;

	// thread id = row index
	long drow = blockDim.x * blockIdx.x + threadIdx.x ;

	long col0val;
	if (drow == 0) {
		col0val = inframe[0];
	}
	else if (drow < m + 1) {
		col0val = inframe[n+1+drow-1];
	}
	__syncthreads();

	// skewed rectangle
	for (dcol = 0; dcol < n + m + 1; ++dcol) {
		//		for (drow = 1; drow < m + 1; ++drow) {
		//drow = thix;
		col = dcol - drow;
		w0 = weftbuff + (dcol % 4)*(m + 1); // % mperiod)*(m + 1); // the current front line of waves
		w1 = weftbuff + ((dcol - 1 + 4) % 4)*(m + 1); // % mperiod)*(m + 1); // the last passed line of waves
		w2 = weftbuff + ((dcol - 2 + 4) % 4)*(m + 1); // % mperiod)*(m + 1); // the second last line of waves
		if (drow == 0) {
			// load the value of the top row from the initial boundary 
			cellval = inframe[col];
		}
		else if (col == 0) {
			// load the value of the left-most column from the initial boundary 
			cellval = col0val;
		}
		else if ((col > 0) && (1 <= drow && drow < m + 1)) {
			ins = w1[drow - 1] + 1;
			del = w1[drow] + 1;
			repl = w2[drow - 1] + (t[col - 1] != p[drow - 1]);
			cellval = ins;
			if (del < cellval)
				cellval = del;
			if (repl < cellval)
				cellval = repl;
		}	
		if (drow < m + 1) {
			//if (cellval < 0 || cellval > 10000)
			//	cellval = -1;
			w0[drow] = cellval;
#ifdef DEBUG_TABLE
			table[(m + 1)*col + drow] = w0[drow];
#endif
			if (drow == m && col <= n)
				outframe[col] = cellval;
			if ( drow < m && col == n ) 
				outframe[n + 1 + drow] = cellval;
		}

		__syncthreads();
	}

	return;
}
