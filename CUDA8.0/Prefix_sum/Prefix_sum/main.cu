
#include "hip/hip_runtime.h"


#include <stdio.h>

static __device__ __forceinline__ unsigned int ntz32_cuda(unsigned int x)
{
	unsigned int ret;
	asm volatile("bfind.u32 %0, %1;" : "=r"(ret) : "r"(x));
	return 31 - ret;
}

__device__ unsigned int ceil2pow32(unsigned int x) {
	if (x == 0)
		return 0;
	return 1 << (32 - ntz32_cuda(x - 1));

}

hipError_t prefixScan(int *a, unsigned const int nsize);

__global__ void prefixKernel(int *a, const int width, const int nsize)
{
    int tidx = threadIdx.x;
	int pow2 = ceil2pow32(width); // 2^k - 1
	if (tidx == 0) {
		printf("pow2 = %d\n", pow2);
	}
	if (tidx < nsize) {
		a[tidx] = a[tidx];
		if ( ((tidx+1) & (pow2 - 1)) == 0) {
			a[tidx] = a[tidx - (pow2 >> 1)] + a[tidx];
		}
		
	}
	else {
		printf("I'm %d, stymied.\n", tidx);
	}
}

int main()
{
    int a[] = { 11, 21, 13, 24, 5 };
	const unsigned int arraySize = sizeof(a)/sizeof(int);

    // Add vectors in parallel.
    hipError_t cudaStatus = prefixScan(a, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "prefscan failed!");
        return 1;
    }

    printf("{ 11, 21, 13, 24, 5 } = {%d,%d,%d,%d,%d}\n",
        a[0], a[1], a[2], a[3], a[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t prefixScan(int *a, unsigned const int nsize) 
{
    int *dev_a = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for vectors a and b.
	cudaStatus = hipMalloc((void**)&dev_a, nsize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, nsize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	for(int width = 2; width < nsize; width <<= 1)
	    prefixKernel<<<1, nsize>>>(dev_a, width, nsize);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "prefixScanKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(a, dev_a, nsize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    
    return cudaStatus;
}
