#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
/*
#include <time.h>
#include <cutil_inline.h>
*/

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#define MATRIX_SIZE 1024/*�s��P�ӂ̐�*/
#define BLOCK_SIZE 16

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv) {
	hipError_t cudaStatus;
	StopWatchInterface *timer = NULL;

	unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;

	int* hMatrixA;
	int* hMatrixB;
	int* hMatrixC;
	hMatrixA = (int*)malloc(matrixSize);
	hMatrixB = (int*)malloc(matrixSize);

	/*�����l�ݒ�*/
	unsigned int col_idx, row_idx;
	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
			hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
			hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
		}
	}

	/*�f�o�C�X���̕ϐ��ݒ�*/
	int* dMatrixA;
	int* dMatrixB;
	int* dMatrixC;

	/*�f�o�C�X�������̈�̊m��*/
	hipMalloc((void**)&dMatrixA, matrixSize);
	hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixB, matrixSize);
	hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&dMatrixC, matrixSize);

	/*�u���b�N�T�C�Y�ƃO���b�h�T�C�Y�̐ݒ�*/
	dim3 block(1, 1);
	dim3 grid(1, 1);

	/*�^�C�}�[���쐬���Čv���J�n*/
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	/*�J�[�l���̋N��*/
	matrixMul << < grid, block >> >(dMatrixA, dMatrixB, dMatrixC);
	hipDeviceSynchronize();

	/*���ʂ̗̈�m�ۂƃf�o�C�X������̃������]��*/
	hMatrixC = (int*)malloc(matrixSize);
	hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost);

	/*�^�C�}�[���~�������������Ԃ�\��*/
	sdkStopTimer(&timer);
	printf("�v�Z���� =%f(ms)\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return EXIT_FAILURE;
	}

	/*�z�X�g�E�f�o�C�X�������̊J��*/
	free(hMatrixA);
	free(hMatrixB);
	free(hMatrixC);
	hipFree(dMatrixA);
	hipFree(dMatrixB);
	hipFree(dMatrixC);

	return EXIT_SUCCESS;
}

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC) {
	unsigned int col_idx;
	unsigned int row_idx;
	unsigned int scan_idx;

	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
			for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
				inMatrixC[col_idx * MATRIX_SIZE + row_idx] += inMatrixA[col_idx * MATRIX_SIZE + scan_idx] *
					inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
			}
		}
	}
}