
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "kernel.h"
#include "cuutils.h"

int main()
{

	cu_test_kernel <<<1, 10>>>();

	CUCHECK(hipGetLastError());

	hipDeviceReset();
    
    return 0;
}
