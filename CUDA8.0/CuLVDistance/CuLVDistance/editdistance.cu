#include "hip/hip_runtime.h"
/*
 * editdistance.c
 *
 *  Created on: 2016/11/26
 *      Author: sin
 */

#include <stdio.h>
#include <stdlib.h>

#include "editdistance.h"

//#define DEBUG_DPTABLE

#define min(x, y)  ((x) <= (y)? (x) : (y))

// assuming the table array size (n+1) x (m+1)
long dptable(long * table, const long colsize, const long rowsize, const char t[], const long n, const char p[], const long m) {
	long dix, dcol, drow; // diagonal column index, row index
	long col; // inner diagonal index
	long result = n+m+1; // an impossible large value
	long ins, del, repl;

#ifdef DEBUG_DPTABLE
	// clear table elements
	for(long r = 0; r < rowsize; ++r)
		for(long c = 0; c < colsize; ++c)
			table[rowsize*c + r] = -1;
#endif

	// initialize
	// do in parallel for each dcol
	for(dcol = 0; dcol <= n; ++dcol) {
		if ( dcol <= m ) {
			dix = dcol*(dcol+1)/2;
		} else {
			dix = m*(m+1)/2+(m+1)*(dcol-m);
		}
		table[dix] = dcol;
	}
	// do in parallel for each drow
	for(drow = 1; drow <= m; ++drow) {
		// m <= n
		dix = (drow+1)*(drow+2)/2 - 1;
		table[dix] = drow;
	}

	// upper-left triangle
	dix = 1;
	for(dcol = 2; dcol <= m + 1; ++dcol) {
		dix += dcol;
		for(drow = 1; drow < dcol; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
#endif
			ins = table[ dix + drow - dcol - 1 ] + 1;
			del = table[ dix + drow - dcol ] + 1;
			repl = table[ dix + drow - 2*dcol]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
			ins = ((ins <= del) ? ins : del);
			table[dix+drow]  = (ins < repl ? ins : repl);
		}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

	// skewed rectangle
	for(dcol = m+2; dcol < n + 1;++dcol) {
		dix += m + 1;
		for(drow = 1; drow < m + 1; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
#endif
			ins = table[ dix + drow - (m+1) - 1] + 1;
			del = table[ dix + drow - (m+1) ] + 1;
			repl = table[ dix + drow - 2 *(m+1) - 1 ]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			ins = ((ins <= del) ? ins : del);
			table[ dix + drow ]  = (ins < repl ? ins : repl);
#ifdef DEBUG_DPTABLE
			//table[(dcol + 2)*rowsize + (drow + 1)] = table[ ((dcol + 2)% (n+2))*rowsize + (drow + 1)];
#endif
			}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

	// bottom-right triangle
	for(dcol = n+1; dcol < n + m + 2; ++dcol) {
		dix += n + m + 1 - dcol;
		for(drow = dcol - n; drow < m+1; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
#endif
			ins = table[ dix + drow - (n + m + 2 - dcol) ] + 1;
			del = table[ dix + drow - (n + m + 2 - dcol) + 1 ] + 1;
			repl = table[ dix + drow - 2*(n + m + 2 - dcol) ]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			//fprintf(stdout, "(%3ld)", n + m + 2 - dcol);
			ins = ((ins <= del) ? ins : del);
			table[ dix + drow ]  = (ins < repl ? ins : repl);
#ifdef DEBUG_DPTABLE
			//table[(dcol + 2)*rowsize + (drow + 1)] = table[ ((dcol + 2)% (n+2))*rowsize + (drow + 1)];
#endif
			}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

#ifdef DEBUG_DPTABLE
	// show DP table
	long r, c;
	for(r = 0; r <= m; r++) {
		for (c = 0; c <= n; c++) {
			if ( c + r <= m ) {
				dix = (c + r)*(c + r + 1)/2 + r;
			} else if (c + r <= n) {
				dix = (m+1)*(m+2)/2 + (m+1)*(c - m - 1 + r) + r;
			} else {
				dix = (m+1)*(m+2)/2 + (m+1)*(c - m - 1 + r) + r - (c + r - n)*(c + r - n + 1)/2;
			}
			fprintf(stdout, "%3ld\t", table[dix]);
		}
		fprintf(stdout, "\n");
	}

	fprintf(stdout, "[%3ld], ", (n+1)*(m+1) - 1 );
#endif
	result = table[ (n+1)*(m+1) - 1 ];
	return result;
}

