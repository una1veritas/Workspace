#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cu_levdist.h"
#include "cu_utils.h"

#include "debug_table.h"

long cu_levdist(long * frame, const char text[], const long n, const char patt[], const long m) {
	long result = n + m + 1;
	long weftlen = n+m+1;

	long * devframe;
	char *devtext, *devpatt;
	hipMalloc((void **)&devframe, sizeof(long)*weftlen);
	hipMemcpy(devframe, frame, sizeof(long)*weftlen, hipMemcpyHostToDevice);
	hipMalloc((void**)&devtext, sizeof(char)*n);
	hipMemcpy(devtext, text, sizeof(char)*n, hipMemcpyHostToDevice);
	hipMalloc((void**)&devpatt, sizeof(char)*m);
	hipMemcpy(devpatt, patt, sizeof(char)*m, hipMemcpyHostToDevice);

#ifdef DEBUG_TABLE
	long * dev_debug_table;
	hipMalloc((void**)&dev_debug_table, sizeof(long)*(n*m));
#endif

	weaving_cdp_kernel <<<1, 1>>> (devframe, weftlen, devtext, n, devpatt, m
#ifdef DEBUG_TABLE
		, dev_debug_table
#endif
		);
	cuCheckErrors(hipGetLastError());

#ifdef DEBUG_TABLE
	hipMemcpy(debug_table, dev_debug_table, sizeof(long)*(n*m),hipMemcpyDeviceToHost);
	hipFree(dev_debug_table);
#endif
	hipMemcpy(frame, devframe, sizeof(long)*weftlen, hipMemcpyDeviceToHost);
	hipFree(devframe);

	cuCheckErrors(hipDeviceSynchronize());

	result = frame[n];

	return result;
}


__global__ void weaving_kernel(long * frame, const long weftlen, const char t[], const long n, const char p[], const long m
#ifdef DEBUG_TABLE
, long * table
#endif
) {
	long col, row;
	long del, ins, repl, cellval; // del = delete from pattern, downward; ins = insert to pattern, rightward
	long warpix, warp_start, warp_last;

	if (frame == NULL)
		return;

	for (long depth = 0; depth <= (n - 1) + (m - 1); depth++) {
		warp_start = abs((m - 1) - depth);
		if (depth < n) {
			warp_last = depth + (m - 1);
		}
		else {
			warp_last = ((n - 1) << 1) + (m - 1) - depth;
		}
		// mywarpix = (thix<<1) + (depth & 1);
		//printf("depth %ld [%ld, %ld]: warpix ", depth, warp_start, warp_last);
		for (long warpix = warp_start; warpix <= warp_last; warpix += 2) {
			if (warpix < 0 || warpix > n + m + 1) {
				printf("warp value error: %ld\n", warpix);
				//fflush(stdout);
			}
			col = (depth + warpix - (m - 1))>>1;
			row = (depth - warpix + (m - 1))>>1;

			//printf("%ld = (%ld, %ld), ", warpix, col, row);
			//
			del = frame[warpix+1+1] + 1;
			ins = frame[warpix-1+1] + 1;
			repl = frame[warpix+1] + (t[col] != p[row]);
			//printf("%ld: %ld [%ld,%ld] %c|%c : %ld/%ld/%ld+%ld,\n",depth, warpix, col,row,t[col],p[row], del,ins, frame[warpix], (t[col] != p[row]));
			//
			if (del < ins) {
				ins = del;
			}
			if (ins < repl) {
				repl = ins;
			}
			//
			frame[warpix+1] = repl;
#ifdef DEBUG_TABLE
			table[m*col + row] = repl;
#endif

		}
		//printf("\n");
		
	}
}


__global__ void weaving_cdp_kernel(long * frame, const long weftlen, const char t[], const long n, const char p[], const long m
#ifdef DEBUG_TABLE
	, long * table
#endif
) {
	long warp_start, warp_last;
	const long threads_per_block = 192;

	if (frame == NULL)
		return;

	//dim3 blocks(ceildiv( (m+1)>>1, threads_per_block)), threads(threads_per_block);
	for (long depth = 0; depth <= (n - 1) + (m - 1); depth++) {
		warp_start = abs((m - 1) - depth);
		if (depth < n) {
			warp_last = depth + (m - 1);
		}
		else {
			warp_last = ((n - 1) << 1) + (m - 1) - depth;
		}
		//printf("depth %ld [%ld, %ld]: warpix ", depth, warp_start, warp_last);
		long warpnum = (warp_last - warp_start + 1)>>1;
		dim3 blocks(ceildiv(warpnum, threads_per_block)), threads(threads_per_block);
		warps_cdp_kernel<<<blocks, threads>>>(frame, t, n, p, m, depth, warp_start, warp_last
#ifdef DEBUG_TABLE
			, table
#endif
		);
		//hipDeviceSynchronize();
		//printf("\n");
	}
}

__global__ void warps_cdp_kernel(long * frame, const char * t, const long n, const char * p, const long m, const long depth, const long warp_start, const long warp_last
#ifdef DEBUG_TABLE
	, long * table
#endif
) {
	long warpix = warp_start + ((blockDim.x * blockIdx.x + threadIdx.x) << 1);
	long del, ins, repl; // del = delete from pattern, downward; ins = insert to pattern, rightward
	long col, row;

	if ( (warp_start <= warpix) && (warpix <= warp_last) ) {
		col = (depth + warpix - (m - 1)) >> 1;
		row = (depth - warpix + (m - 1)) >> 1;

		del = frame[warpix + 2] + 1;
		ins = frame[warpix] + 1;
		repl = frame[warpix + 1] + (t[col] != p[row]);
		//printf("%ld: %ld [%ld,%ld] %c|%c : %ld/%ld/%ld+%ld,\n",depth, warpix, col,row,t[col],p[row], del,ins, frame[warpix], (t[col] != p[row]));
		//
		if (del < ins) {
			ins = del;
		}
		if (ins < repl) {
			repl = ins;
		}
		//
		frame[warpix + 1] = repl;
#ifdef DEBUG_TABLE
		table[m*col + row] = repl;
#endif
	}
	__syncthreads();
}
