#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string.h>

#include <helper_timer.h>

#include "cu_utils.h"
#include "levdist.h"
#include "cu_levdist.h"
#include "textfromfile.h"

#include "debug_table.h"

#define MEGA_B 1048576UL
#define KILO_B 1024UL
#define STR_MAXLENGTH (32 * KILO_B)

int getargs(const int argc, const char * argv[], char * text, char * patt, long * n, long *m) {
	if ( argc != 3 )
		return EXIT_FAILURE;

	text[STR_MAXLENGTH - 1] = 0;
	patt[STR_MAXLENGTH - 1] = 0;

	if ( textfromfile(argv[1], STR_MAXLENGTH, text) != 0
		|| textfromfile(argv[2], STR_MAXLENGTH, patt) != 0 ) {
		return EXIT_FAILURE;
	}
	*n = (text[STR_MAXLENGTH-1] == 0? strlen(text) : STR_MAXLENGTH);
	*m = (patt[STR_MAXLENGTH-1] == 0? strlen(patt) : STR_MAXLENGTH);
	if ( *n < *m ) {
		char * tmp = text;
		text = patt;
		patt = tmp;
		long t = *n;
		*n = *m;
		*m = t;
	}

	if ( *n < 1000 && *m < 1000 )
		fprintf(stdout, "Input: %s \n(%lu), \n%s \n(%lu)\n\n", text, *n, patt, *m);
	else
		fprintf(stdout, "Input: (%lu), (%lu)\n\n", *n, *m);
	fflush(stdout);

	return 0;
}

int main(int argc, const char * argv[]) {
	char * text, *patt;
	long * table;
	long m, n;
	long d;

	hipSetDevice(0);
	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 2);

	text = (char*)malloc(sizeof(char)*STR_MAXLENGTH);
	patt = (char*)malloc(sizeof(char)*STR_MAXLENGTH);
	if (text == NULL || patt == NULL) {
		fprintf(stderr, "malloc error.\n");
		fflush(stderr);
		goto exit_error;
	}


	if (getargs(argc, argv, text, patt, &n, &m) != 0)
		goto exit_error;

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);

#ifdef USE_PURE_DP
	sdkStartTimer(&timer);

	table = (long*)malloc(sizeof(long)*m*n);

	d = dp_edist(table, text, n, patt, m);
#ifndef DEBUG_TABLE
	free(table);
#endif

	sdkStopTimer(&timer);
	printf("\nElapsed %f msec.\n", sdkGetTimerValue(&timer));

	printf("Edit distance (by Pure DP): %lu\n", d);
#ifdef DEBUG_TABLE
	if (max(n, m) < 128)
		show_table(table, n, m);

	debug_table = (long*)malloc(sizeof(long)*m*n);
#endif
#endif USE_PURE_DP

	fprintf(stdout, "\nNow computing edit distance by Weaving DP.\n");
	fflush(stdout);

	long * frame = (long*)malloc(sizeof(long)*(m + n + 1));
	weaving_setframe(frame, n, m);

	printf("frame input: \n");
	for (int i = 0; i < min(n + m + 1, 64); i++) {
		printf("%d, ", frame[i]);
	}
	printf("\n");
	fflush(stdout);

	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	d = cu_levdist(frame, text, n, patt, m);

	sdkStopTimer(&timer);
	printf("\nElapsed %f msec.\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	printf("Edit distance (by Weaving DP): %lu\n\n", d);
	printf("frame output: \n");
	for (int i = 0; i < min(n + m + 1, 64); i++) {
		printf("%d, ", frame[i]);
	}
	printf("\n");
	fflush(stdout);
	free(frame);

#ifdef DEBUG_TABLE
	if ( max(n,m) < 128 )
		show_table(debug_table, n, m);
	if ( compare_table(debug_table, table, n, m) != 0) {
		printf("table compare failed.\n");
	} else {
		printf("two tables are identical.\n");
	}
	free(debug_table);
	free(table);
#endif

exit_error:
	free(text);
	free(patt);

    return 0;
}

