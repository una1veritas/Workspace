#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define NX      256
#define BATCH   10

int main(int argc, char *argv[])
{
	hipfftHandle plan;
	hipfftComplex *devPtr;
	hipfftComplex data[NX*BATCH];
	int i;

	
	printf("[simpleCUFFT] is starting...\n");
	findCudaDevice(argc, (const char **)argv);
	
	if (argc != 3)
		return EXIT_FAILURE;

	/* ���f�[�^�쐬 */
	for (i = 0; i < NX*BATCH; i++) {
		data[i].x = 1.0f;
		data[i].y = 1.0f;
	}

	/* GPU�p���������蓖�� */
	hipMalloc((void**)&devPtr, sizeof(hipfftComplex)*NX*BATCH);

	/* GPU�p�������ɓ]�� */
	hipMemcpy(devPtr, data, sizeof(hipfftComplex)*NX*BATCH, hipMemcpyHostToDevice);

	/* 1D FFT plan�쐬 */
	hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);

	/* FFT�������{ */
	hipfftExecC2C(plan, devPtr, devPtr, HIPFFT_FORWARD);

	/* FFT�������{(�t�ϊ�) */
	/*
	hipfftExecC2C(plan, devPtr, devPtr, HIPFFT_BACKWARD);
	*/

	/* �v�Z���ʂ�GPU����������]�� */
	hipMemcpy(data, devPtr, sizeof(hipfftComplex)*NX*BATCH, hipMemcpyDeviceToHost);

	/* CUFFT plan�폜 */
	hipfftDestroy(plan);

	/* GPU�p�������J�� */
	hipFree(devPtr);

	for (i = 0; i < NX*BATCH; i++) {
		printf("data[%d] %f %f\n", i, data[i].x, data[i].y);
	}

	return 0;
}