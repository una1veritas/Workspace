#include "hip/hip_runtime.h"

/*
 * editdistance.c
 *
 *  Created on: 2016/11/26
 *      Author: sin
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "editdistance.h"

#define MAX_THREADSPERBLOCK 1024

#define DEBUG_DPTABLE

#define min(x, y)  ((x) <= (y)? (x) : (y))
#define max(x, y)  ((x) >= (y)? (x) : (y))


static char grayscale[] = "$@%&M#*oahkbdpqwmZOQLCJUYXzcvunxrjft/\|()1{]?-_+~<>i!lI;:,\"^`'. ";

long alignval(const long base, const long val) {
	return ((val + base - 1) / val)*val;
}

int cuStatCheck(const hipError_t stat, const char * msg) {
	if (stat != hipSuccess) {
		fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(stat));
		return -1;
	}
	return 0;
}

long cu_lvdist(long * table, const char t[], const long n, const char p[], const long m) {
	long dix;
	long result = n + m + 1; // an impossible value

	hipError_t cuStat;

#ifdef DEBUG_DPTABLE
	long r, c;
	// clear table elements
	for (long r = 0; r < (m + 1); ++r)
		for (long c = 0; c < (n + 1); ++c)
			table[(m + 1)*c + r] = -1;
#endif

	char * devt, *devp;
	cuStat = hipMalloc((void**) &devt, n);
	hipMemcpy(devt, t, n, hipMemcpyHostToDevice);
	cuStatCheck(cuStat, "hipMalloc devt");
	cuStat = hipMalloc((void**) &devp, m);
	hipMemcpy(devp, p, m, hipMemcpyHostToDevice);
	cuStatCheck(cuStat, "hipMalloc devp");

	long *devboundary;
	hipMalloc((void**)&devboundary, sizeof(long)*alignval(32,n + 1 + m));

	long * devtable;
	const long tablesize = alignval(32, n + m + 1)*(m + 1);
	cuStat = hipMalloc((void**)&devtable, sizeof(long)*tablesize);
	cuStatCheck(cuStat, "hipMalloc devtable failed.\n");
	//hipMemcpy(devtable, table, tablesize , hipMemcpyHostToDevice);

	fprintf(stdout, "copied input, calling kernel...\n");
	fflush(stdout);

	long nthreads = alignval(32, m+1);
	dim3 grids(max(1, nthreads / MAX_THREADSPERBLOCK), 1), blocks(MAX_THREADSPERBLOCK);
	fprintf(stdout,"num threads %d, %d blocks.\n",nthreads, max(1, nthreads / MAX_THREADSPERBLOCK));

	cu_init_row << <grids, blocks >> >(devboundary, n, m);

	cu_dptable<<< grids, blocks >>>(devtable, devboundary, devt, n, devp, m);

	// Check for any errors launching the kernel
	cuStat = hipGetLastError();
	if (cuStat != hipSuccess) {
		fprintf(stderr, "kernel function(s) failed: %s\n", hipGetErrorString(cuStat));
	}
	fprintf(stdout,"Finished kernel functions.\n");
	fflush(stdout);

	//hipMemcpy(table, devtable, tablesize, hipMemcpyDeviceToHost);
	hipMemcpy(table, devtable, sizeof(long)*(n+m+1)*(m+1), hipMemcpyDeviceToHost);
	hipFree(devtable);

#ifdef DEBUG_DPTABLE
	// show DP table
	for (r = 0; r < m+1; r++) {
		for (c = 0; c < n+1; c++) {
			dix = (m + 1)*(c + r) + r;
			fprintf(stdout, "%c ", grayscale[min(63,table[dix]*64/m)] );
			/*
			if (n > 40 && c == 32) {
				c = n - 6;
				fprintf(stdout, " ... ");
			}
			*/
		}
		fprintf(stdout, "\n");
	}
#endif

	result = table[(n+1)*(m+1) - 1];

	return result;
}

__global__ void cu_init_row(long * row, const long n, const long m) {
	long thix = blockDim.x * blockIdx.x + threadIdx.x;

	if (thix < n+1) {
		row[thix] = thix;
	} 
	if (thix < m) {
		row[n+2+thix] = thix+1;
	}
	__syncthreads();
}

// assuming the table array size (n+1) x (m+1)
__global__ void cu_dptable(long * table, const long * boundary, const char t[], const long n, const char p[], const long m) {
	long dix, dcol, drow; // diagonal column index, row index
	long col; // inner diagonal index
	long ins, del, diff, repl;

	long thix = blockDim.x * blockIdx.x + threadIdx.x ;

	/*
#ifdef DEBUG_DPTABLE
	if (thix < (n+1) * (m + 1))
		table[thix] = 0;
	__syncthreads();
#endif
*/

	// skewed rectangle
	for (dcol = 0, dix = 0; dcol < n + m + 1; ++dcol, dix += m+1) {
		//		for (drow = 1; drow < m + 1; ++drow) {
		drow = thix;
		col = dcol - drow;
		if (drow == 0) {
			table[dix] = boundary[col];
		} else if (col == 0) { // drow != 0
			table[dix + drow] = boundary[n + 1 + drow];
		} else if ( (col > 0) && (1 <= drow && drow < m+1) ) {
			ins = table[dix + drow - (m + 1) - 1] + 1;
			del = table[dix + drow - (m + 1)] + 1;
			diff = 0;
			if (t[col - 1] != p[drow - 1])
				diff = 1;
			repl = table[dix + drow - 2 * (m + 1) - 1] + diff;
			if (ins > del)
				ins = del;
			if (repl > ins)
				repl = ins;
			table[dix + drow] = repl;
		}
		__syncthreads();
	}

	return;

	// bottom-right triangle
	for (dcol = n + 1; dcol < n + m + 2; ++dcol) {
		dix += n + m + 1 - dcol;
//		for (drow = dcol - n; drow < m + 1; ++drow) {
		drow = thix;
		if ( drow >= dcol - n && drow < m + 1 ) {
			col = dcol - drow;
			ins = table[dix + drow - (n + m + 2 - dcol)] + 1;
			del = table[dix + drow - (n + m + 2 - dcol) + 1] + 1;
			diff = 0;
			if (t[col - 1] != p[drow - 1])
				diff = 1;
			repl = table[dix + drow - 2 * (n + m + 2 - dcol)] + diff;
			//fprintf(stdout, "(%3ld)", n + m + 2 - dcol);
			if (ins > del)
				ins = del;
			if (repl > ins)
				repl = ins;
			table[dix + drow] = repl;
		}
		__syncthreads();
	}
	
	return;
}


long lvdist(long * table, const char t[], const long n, const char p[], const long m) {
	long dix, dcol, drow; // diagonal column index, row index
	long col; // inner diagonal index
	long result = n+m+1; // an impossible large value
	long ins, del, repl;

#ifdef DEBUG_DPTABLE
	// clear table elements
	for(long r = 0; r < (m+1); ++r)
		for(long c = 0; c < (n+1); ++c)
			table[(m+1)*c + r] = -1;
#endif

	// initialize
	// do in parallel for each dcol
	for(dcol = 0; dcol <= n; ++dcol) {
		if ( dcol <= m ) {
			dix = dcol*(dcol+1)/2;
		} else {
			dix = m*(m+1)/2+(m+1)*(dcol-m);
		}
		table[dix] = dcol;
	}
	// do in parallel for each drow
	for(drow = 1; drow <= m; ++drow) {
		// m <= n
		dix = (drow+1)*(drow+2)/2 - 1;
		table[dix] = drow;
	}

	// upper-left triangle
	dix = 1;
	for(dcol = 2; dcol <= m + 1; ++dcol) {
		dix += dcol;
		for(drow = 1; drow < dcol; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
#endif
			ins = table[ dix + drow - dcol - 1 ] + 1;
			del = table[ dix + drow - dcol ] + 1;
			repl = table[ dix + drow - 2*dcol]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
			ins = ((ins <= del) ? ins : del);
			table[dix+drow]  = (ins < repl ? ins : repl);
		}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

	// skewed rectangle
	for(dcol = m+2; dcol < n + 1;++dcol) {
		dix += m + 1;
		for(drow = 1; drow < m + 1; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
#endif
			ins = table[ dix + drow - (m+1) - 1] + 1;
			del = table[ dix + drow - (m+1) ] + 1;
			repl = table[ dix + drow - 2 *(m+1) - 1 ]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			ins = ((ins <= del) ? ins : del);
			table[ dix + drow ]  = (ins < repl ? ins : repl);
#ifdef DEBUG_DPTABLE
			//table[(dcol + 2)*rowsize + (drow + 1)] = table[ ((dcol + 2)% (n+2))*rowsize + (drow + 1)];
#endif
			}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

	// bottom-right triangle
	for(dcol = n+1; dcol < n + m + 2; ++dcol) {
		dix += n + m + 1 - dcol;
		for(drow = dcol - n; drow < m+1; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
#endif
			ins = table[ dix + drow - (n + m + 2 - dcol) ] + 1;
			del = table[ dix + drow - (n + m + 2 - dcol) + 1 ] + 1;
			repl = table[ dix + drow - 2*(n + m + 2 - dcol) ]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			//fprintf(stdout, "(%3ld)", n + m + 2 - dcol);
			ins = ((ins <= del) ? ins : del);
			table[ dix + drow ]  = (ins < repl ? ins : repl);
#ifdef DEBUG_DPTABLE
			//table[(dcol + 2)*rowsize + (drow + 1)] = table[ ((dcol + 2)% (n+2))*rowsize + (drow + 1)];
#endif
			}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

#ifdef DEBUG_DPTABLE
	// show DP table
	long r, c;
	for(r = 0; r <= m; r++) {
		for (c = 0; c <= n; c++) {
			if ( c + r <= m ) {
				dix = (c + r)*(c + r + 1)/2 + r;
			} else if (c + r <= n) {
				dix = (m+1)*(m+2)/2 + (m+1)*(c - m - 1 + r) + r;
			} else {
				dix = (m+1)*(m+2)/2 + (m+1)*(c - m - 1 + r) + r - (c + r - n)*(c + r - n + 1)/2;
			}
			fprintf(stdout, "%3ld\t", table[dix]);
		}
		fprintf(stdout, "\n");
	}

	fprintf(stdout, "[%3ld], ", (n+1)*(m+1) - 1 );
#endif
	result = table[ (n+1)*(m+1) - 1 ];
	return result;
}

