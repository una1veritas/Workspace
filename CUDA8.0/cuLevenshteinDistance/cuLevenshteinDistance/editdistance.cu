#include "hip/hip_runtime.h"

/*
 * editdistance.c
 *
 *  Created on: 2016/11/26
 *      Author: sin
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "editdistance.h"

#define MAX_THREADSPERBLOCK 1024

//#define DEBUG_DPTABLE

#define min(x, y)  ((x) <= (y)? (x) : (y))
#define max(x, y)  ((x) >= (y)? (x) : (y))

long pow2(long val) {
	long result;
	for (result = 1; result < val; result <<= 1);
	return result;
}

int cuStatCheck(const hipError_t stat, const char * msg) {
	if (stat != hipSuccess) {
		fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(stat));
		return -1;
	}
	return 0;
}

long cu_lvdist(long * table, const char t[], const long n, const char p[], const long m) {
	long result = n + m + 1; // an impossible value

	hipError_t cuStat;

#ifdef DEBUG_DPTABLE
	long r, c;
	// clear table elements
	for (long r = 0; r < (m + 1); ++r)
		for (long c = 0; c < (n + 1); ++c)
			table[(m + 1)*c + r] = -1;
#endif

	char * devt, *devp;
	cuStat = hipMalloc((void**) &devt, n);
	cuStatCheck(cuStat, "hipMalloc devt");

	cuStat = hipMalloc((void**) &devp, m);
	cuStatCheck(cuStat, "hipMalloc devp");
	hipMemcpy(devt, t, n, hipMemcpyHostToDevice);
	hipMemcpy(devp, p, m, hipMemcpyHostToDevice);

	const long tablesize = sizeof(long) * (n + 1)*(m + 1);
	long * devtable;
	cuStat = hipMalloc((void**)&devtable, tablesize);
	cuStatCheck(cuStat, "hipMalloc devtable failed.\n");
	hipMemcpy(devtable, table, tablesize , hipMemcpyHostToDevice);

	fprintf(stdout, "copied input, calling kernel...\n");
	fflush(stdout);

	long nthreads = pow2(max(n+1,m+1));
	fprintf(stdout,"num threads %d, %d grids.\n",nthreads, max(1, nthreads / MAX_THREADSPERBLOCK));
	fflush(stdout);
	//cu_dptable <<< max(1, nthreads / MAX_THREADSPERBLOCK), MAX_THREADSPERBLOCK >>> (devtable, devt, n, devp, m);

	long dix, dcol;
	// init top and left-most lines
	cu_dptable_init << < max(1, nthreads / MAX_THREADSPERBLOCK), MAX_THREADSPERBLOCK >> > (devtable, devt, n, devp, m);
	// upper-left triangle
	dix = 1;
	for (dcol = 2; dcol <= m + 1; ++dcol) {
		dix += dcol;
		cu_dptable_topleft<< < max(1, nthreads / MAX_THREADSPERBLOCK), MAX_THREADSPERBLOCK >> > (devtable, devt, n, devp, m, dix, dcol);
	}
	// skewed rectangle
	for (dcol = m + 2; dcol < n + 1; ++dcol) {
		dix += m + 1;
		cu_dptable_center << < max(1, nthreads / MAX_THREADSPERBLOCK), MAX_THREADSPERBLOCK >> > (devtable, devt, n, devp, m, dix, dcol);
	}
	// bottom-right triangle
	for (dcol = n + 1; dcol < n + m + 2; ++dcol) {
		dix += n + m + 1 - dcol;
		cu_dptable_bottomright << < max(1, nthreads / MAX_THREADSPERBLOCK), MAX_THREADSPERBLOCK >> > (devtable, devt, n, devp, m, dix, dcol);
	}
	// end of the dptable computaton

	// Check for any errors launching the kernel
	cuStat = hipGetLastError();
	if (cuStat != hipSuccess) {
		fprintf(stderr, "kernel function(s) failed: %s\n", hipGetErrorString(cuStat));
	}
	fprintf(stdout,"Finished kernel functions.\n");
	fflush(stdout);

	hipMemcpy(table, devtable, tablesize, hipMemcpyDeviceToHost);
	hipFree(devtable);

#ifdef DEBUG_DPTABLE
	// show DP table
	for (r = 0; r <= m; r++) {
		for (c = 0; c <= n; c++) {
			if (c + r <= m) {
				dix = (c + r)*(c + r + 1) / 2 + r;
			}
			else if (c + r <= n) {
				dix = (m + 1)*(m + 2) / 2 + (m + 1)*(c - m - 1 + r) + r;
			}
			else {
				dix = (m + 1)*(m + 2) / 2 + (m + 1)*(c - m - 1 + r) + r - (c + r - n)*(c + r - n + 1) / 2;
			}
			fprintf(stdout, "%4ld ", table[dix]);
			/*
			if (n > 40 && c == 32) {
				c = n - 6;
				fprintf(stdout, " ... ");
			}
			*/
		}
		fprintf(stdout, "\n");
	}
#endif

	result = table[(n+1)*(m+1) - 1];

	return result;
}

// assuming the table array size (n+1) x (m+1)
__global__ void cu_dptable_by1block(long * table, const char t[], const long n, const char p[], const long m) {
	long dix, dcol, drow; // diagonal column index, row index
	long col; // inner diagonal index
	long ins, del, diff, repl;

	long thix = blockDim.x * blockIdx.x + threadIdx.x ;

	
#ifdef DEBUG_DPTABLE
	if (thix < (n+1) * (m + 1))
		table[thix] = 0;
	__syncthreads();
#endif

	// initialize
	// do in parallel for each dcol
	// for (dcol = 0; dcol <= n; ++dcol) {
	dcol = thix;
	if (dcol <= m) {
		dix = dcol*(dcol + 1) / 2;
	}
	else if (dcol <= n ) {
		dix = m*(m + 1) / 2 + (m + 1)*(dcol - m);
	}
	if ( dcol <= n )
		table[dix] = dcol;
	__syncthreads();

	
	// do in parallel for each drow
	// for (drow = 1; drow <= m; ++drow) {
	drow = thix;
	if ( drow > 0 && drow <= m ) {
		// m <= n
		dix = (drow + 1)*(drow + 2) / 2 - 1;
		table[dix] = drow;
	}
	__syncthreads();


	// upper-left triangle
	dix = 1;
	for (dcol = 2; dcol <= m + 1; ++dcol) {
		dix += dcol;
//		for (drow = 1; drow < dcol; ++drow) {
		drow = thix;
		if ( drow >= 1 && drow < dcol ) {
			col = dcol - drow;
			ins = table[dix + drow - dcol - 1] + 1;
			del = table[dix + drow - dcol] + 1;
			diff = 0;
			if (t[col - 1] != p[drow - 1])
				diff = 1;
			repl = table[dix + drow - 2 * dcol]	+ diff;
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
			if ( ins > del )
				ins = del;
			if (repl > ins)
				repl = ins;
			table[dix + drow] = repl;
		}	
		__syncthreads();
	}
	

	// skewed rectangle
	for (dcol = m + 2; dcol < n + 1; ++dcol) {
		dix += m + 1;
//		for (drow = 1; drow < m + 1; ++drow) {
		drow = thix;
		if ( drow >= 1 && drow < m+1 ) {
			col = dcol - drow;
			ins = table[dix + drow - (m + 1) - 1] + 1;
			del = table[dix + drow - (m + 1)] + 1;
			diff = 0;
			if (t[col - 1] != p[drow - 1])
				diff = 1;
			repl = table[dix + drow - 2 * (m + 1) - 1] + diff;
			if (ins > del)
				ins = del;
			if (repl > ins)
				repl = ins;
			table[dix + drow] = repl;
		}
	__syncthreads();
	}

	
	// bottom-right triangle
	for (dcol = n + 1; dcol < n + m + 2; ++dcol) {
		dix += n + m + 1 - dcol;
//		for (drow = dcol - n; drow < m + 1; ++drow) {
		drow = thix;
		if ( drow >= dcol - n && drow < m + 1 ) {
			col = dcol - drow;
			ins = table[dix + drow - (n + m + 2 - dcol)] + 1;
			del = table[dix + drow - (n + m + 2 - dcol) + 1] + 1;
			diff = 0;
			if (t[col - 1] != p[drow - 1])
				diff = 1;
			repl = table[dix + drow - 2 * (n + m + 2 - dcol)] + diff;
			//fprintf(stdout, "(%3ld)", n + m + 2 - dcol);
			if (ins > del)
				ins = del;
			if (repl > ins)
				repl = ins;
			table[dix + drow] = repl;
		}
		__syncthreads();
	}
	
	return;
}

__global__ void cu_dptable_init(long * table, const char t[], const long n, const char p[], const long m) {
	long dix, dcol, drow; // diagonal column index, row index
	long col; // inner diagonal index
	long ins, del, diff, repl;

	long thix = blockDim.x * blockIdx.x + threadIdx.x;


#ifdef DEBUG_DPTABLE
	if (thix < (n + 1) * (m + 1))
		table[thix] = 0;
	__syncthreads();
#endif

	// initialize
	// do in parallel for each dcol
	// for (dcol = 0; dcol <= n; ++dcol) {
	dcol = thix;
	if (dcol <= m) {
		dix = dcol*(dcol + 1) / 2;
	}
	else if (dcol <= n) {
		dix = m*(m + 1) / 2 + (m + 1)*(dcol - m);
	}
	if (dcol <= n)
		table[dix] = dcol;

	// do in parallel for each drow
	// for (drow = 1; drow <= m; ++drow) {
	drow = thix;
	if (drow > 0 && drow <= m) {
		// m <= n
		dix = (drow + 1)*(drow + 2) / 2 - 1;
		table[dix] = drow;
	}
	//__syncthreads();

	return;
}

__global__ void cu_dptable_topleft(long * table, const char t[], const long n, const char p[], const long m, long dix, long dcol) {
	long col; // inner diagonal index
	long ins, del, diff, repl;

	long drow = blockDim.x * blockIdx.x + threadIdx.x; // thix

	//for (drow = 1; drow < dcol; ++drow) {
	if (drow >= 1 && drow < dcol) {
		col = dcol - drow;
		ins = table[dix + drow - dcol - 1] + 1;
		del = table[dix + drow - dcol] + 1;
		diff = 0;
		if (t[col - 1] != p[drow - 1])
			diff = 1;
		repl = table[dix + drow - 2 * dcol] + diff;
		//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
		if (ins > del)
			ins = del;
		if (repl > ins)
			repl = ins;
		table[dix + drow] = repl;
	}
	//__syncthreads();

	return;
}

__global__ void cu_dptable_center(long * table, const char t[], const long n, const char p[], const long m, const long dix, const long dcol) {
	long col; // inner diagonal index
	long ins, del, diff, repl;

	long drow = blockDim.x * blockIdx.x + threadIdx.x;

	// skewed rectangle
	//		for (drow = 1; drow < m + 1; ++drow) {
	if (drow >= 1 && drow < m + 1) {
		col = dcol - drow;
		ins = table[dix + drow - (m + 1) - 1] + 1;
		del = table[dix + drow - (m + 1)] + 1;
		diff = 0;
		if (t[col - 1] != p[drow - 1])
			diff = 1;
		repl = table[dix + drow - 2 * (m + 1) - 1] + diff;
		if (ins > del)
			ins = del;
		if (repl > ins)
			repl = ins;
		table[dix + drow] = repl;
	}
	//__syncthreads();

	return;
}

__global__ void cu_dptable_bottomright(long * table, const char t[], const long n, const char p[], const long m, const long dix, const long dcol) {
	long col; // inner diagonal index
	long ins, del, diff, repl;

	long drow = blockDim.x * blockIdx.x + threadIdx.x;

	// bottom-right triangle
		//		for (drow = dcol - n; drow < m + 1; ++drow) {
	if (drow >= dcol - n && drow < m + 1) {
		col = dcol - drow;
		ins = table[dix + drow - (n + m + 2 - dcol)] + 1;
		del = table[dix + drow - (n + m + 2 - dcol) + 1] + 1;
		diff = 0;
		if (t[col - 1] != p[drow - 1])
			diff = 1;
		repl = table[dix + drow - 2 * (n + m + 2 - dcol)] + diff;
		//fprintf(stdout, "(%3ld)", n + m + 2 - dcol);
		if (ins > del)
			ins = del;
		if (repl > ins)
			repl = ins;
		table[dix + drow] = repl;
	}
	//__syncthreads();

	return;
}


long lvdist(long * table, const char t[], const long n, const char p[], const long m) {
	long dix, dcol, drow; // diagonal column index, row index
	long col; // inner diagonal index
	long result = n+m+1; // an impossible large value
	long ins, del, repl;

#ifdef DEBUG_DPTABLE
	// clear table elements
	for(long r = 0; r < (m+1); ++r)
		for(long c = 0; c < (n+1); ++c)
			table[(m+1)*c + r] = -1;
#endif

	// initialize
	// do in parallel for each dcol
	for(dcol = 0; dcol <= n; ++dcol) {
		if ( dcol <= m ) {
			dix = dcol*(dcol+1)/2;
		} else {
			dix = m*(m+1)/2+(m+1)*(dcol-m);
		}
		table[dix] = dcol;
	}
	// do in parallel for each drow
	for(drow = 1; drow <= m; ++drow) {
		// m <= n
		dix = (drow+1)*(drow+2)/2 - 1;
		table[dix] = drow;
	}

	// upper-left triangle
	dix = 1;
	for(dcol = 2; dcol <= m + 1; ++dcol) {
		dix += dcol;
		for(drow = 1; drow < dcol; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
#endif
			ins = table[ dix + drow - dcol - 1 ] + 1;
			del = table[ dix + drow - dcol ] + 1;
			repl = table[ dix + drow - 2*dcol]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
			ins = ((ins <= del) ? ins : del);
			table[dix+drow]  = (ins < repl ? ins : repl);
		}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

	// skewed rectangle
	for(dcol = m+2; dcol < n + 1;++dcol) {
		dix += m + 1;
		for(drow = 1; drow < m + 1; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
			//fprintf(stdout, " %c=%c? ",t[col-1],p[drow-1] );
#endif
			ins = table[ dix + drow - (m+1) - 1] + 1;
			del = table[ dix + drow - (m+1) ] + 1;
			repl = table[ dix + drow - 2 *(m+1) - 1 ]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			ins = ((ins <= del) ? ins : del);
			table[ dix + drow ]  = (ins < repl ? ins : repl);
#ifdef DEBUG_DPTABLE
			//table[(dcol + 2)*rowsize + (drow + 1)] = table[ ((dcol + 2)% (n+2))*rowsize + (drow + 1)];
#endif
			}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

	// bottom-right triangle
	for(dcol = n+1; dcol < n + m + 2; ++dcol) {
		dix += n + m + 1 - dcol;
		for(drow = dcol - n; drow < m+1; ++drow) {
			col = dcol - drow;
#ifdef DEBUG_DPTABLE
			fprintf(stdout, "%3ld:%3ld [%2ld, %2ld] ", dcol, dix+drow, col, drow);
#endif
			ins = table[ dix + drow - (n + m + 2 - dcol) ] + 1;
			del = table[ dix + drow - (n + m + 2 - dcol) + 1 ] + 1;
			repl = table[ dix + drow - 2*(n + m + 2 - dcol) ]
						  + (t[col-1] == p[drow-1] ? 0 : 1);
			//fprintf(stdout, "(%3ld)", n + m + 2 - dcol);
			ins = ((ins <= del) ? ins : del);
			table[ dix + drow ]  = (ins < repl ? ins : repl);
#ifdef DEBUG_DPTABLE
			//table[(dcol + 2)*rowsize + (drow + 1)] = table[ ((dcol + 2)% (n+2))*rowsize + (drow + 1)];
#endif
			}
#ifdef DEBUG_DPTABLE
		fprintf(stdout, "\n");
#endif
	}
#ifdef DEBUG_DPTABLE
	fprintf(stdout, "\n");
#endif

#ifdef DEBUG_DPTABLE
	// show DP table
	long r, c;
	for(r = 0; r <= m; r++) {
		for (c = 0; c <= n; c++) {
			if ( c + r <= m ) {
				dix = (c + r)*(c + r + 1)/2 + r;
			} else if (c + r <= n) {
				dix = (m+1)*(m+2)/2 + (m+1)*(c - m - 1 + r) + r;
			} else {
				dix = (m+1)*(m+2)/2 + (m+1)*(c - m - 1 + r) + r - (c + r - n)*(c + r - n + 1)/2;
			}
			fprintf(stdout, "%3ld\t", table[dix]);
		}
		fprintf(stdout, "\n");
	}

	fprintf(stdout, "[%3ld], ", (n+1)*(m+1) - 1 );
#endif
	result = table[ (n+1)*(m+1) - 1 ];
	return result;
}

