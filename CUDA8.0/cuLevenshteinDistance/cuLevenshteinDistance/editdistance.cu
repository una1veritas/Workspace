#include "hip/hip_runtime.h"

/*
 * editdistance.c
 *
 *  Created on: 2016/11/26
 *      Author: sin
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define DEBUG_DPTABLE

#include "editdistance.h"

#define MAX_THREADSPERBLOCK 1024

#define min(x, y)  ((x) <= (y)? (x) : (y))
#define max(x, y)  ((x) >= (y)? (x) : (y))
#define align(base, val)    ((((val)+(base)-1)/(base))*(base))

static char grayscale[] = "@#$B%8&WM*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\|()1{}[]?-_+~<>ilI;!:,\"^`'. ";

int cuStatCheck(const hipError_t stat, const char * msg) {
	if (stat != hipSuccess) {
		fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(stat));
		fflush(stderr);
		return -1;
	}
	return 0;
}


long cu_lvdist(long * inbound, long * outbound, const char t[], const long n, const char p[], const long m) {
	long result = n + m + 1; // an impossible value

	hipError_t cuStat;

	char * devt, *devp;
	long * devwavebuff, *devframe;

	cuStat = hipMalloc((void**) &devt, n);
	hipMemcpy(devt, t, n, hipMemcpyHostToDevice);
	cuStatCheck(cuStat, "hipMalloc devt");
	cuStat = hipMalloc((void**) &devp, m);
	hipMemcpy(devp, p, m, hipMemcpyHostToDevice);
	cuStatCheck(cuStat, "hipMalloc devp");


	long * devweftbuff, * devinframe, *devoutframe;
	const long table_height = m + 1;
	const long table_width = n + m + 1;

	cuStat = hipMalloc((void**)&devweftbuff, sizeof(long)*table_height*4);
	cuStatCheck(cuStat, "hipMalloc devtable failed.\n");

	long * dptable;
	long * devtable;
#ifdef DEBUG_DPTABLE
	cuStat = hipMalloc((void**)&devtable, sizeof(long)*table_height*table_width);
	cuStatCheck(cuStat, "hipMalloc devtable failed.\n");
#endif
	cuStat = hipMalloc((void**)&devinframe, sizeof(long)*(n + m + 1));
	cuStatCheck(cuStat, "hipMalloc devinframe failed.\n");
	cuStat = hipMalloc((void**)&devoutframe, sizeof(long)*(n + 1 + m));
	cuStatCheck(cuStat, "hipMalloc devoutframe failed.\n");
	hipMemcpy(devinframe, inbound, sizeof(long)*(n + m + 1), hipMemcpyHostToDevice);

	fprintf(stdout, "copied input, calling kernel...\n");
	fflush(stdout);

	long nthreads = align(32, m+1);
	dim3 grids(max(1, nthreads / MAX_THREADSPERBLOCK), 1), blocks(MAX_THREADSPERBLOCK);
	fprintf(stdout,"num threads %d, %d blocks.\n",nthreads, max(1, nthreads / MAX_THREADSPERBLOCK));

	cu_dptable<<< grids, blocks >>>(devweftbuff, devinframe, devoutframe, devt, n, devp, m, devtable);

	// Check for any errors launching the kernel
	cuStat = hipGetLastError();
	if (cuStat != hipSuccess) {
		fprintf(stderr, "kernel function(s) failed: %s\n", hipGetErrorString(cuStat));
	}
	fprintf(stdout,"Finished kernel functions.\n");
	fflush(stdout);

	hipMemcpy(outbound, devoutframe, sizeof(long)*(n + m + 1), hipMemcpyDeviceToHost);

#ifdef DEBUG_DPTABLE
	long * table;
	table = (long*)malloc(sizeof(long)*table_height*table_width);
	hipMemcpy(table, devtable, sizeof(long)*table_height*table_width, hipMemcpyDeviceToHost);
	// show DP table
	long c, r, dix;
	const long scales = strlen(grayscale) - 1;
	for (r = 0; r < m + 1; r++) {
		for (c = 0; c < n + 1; c++) {
			long gray = m - table[c*(m+1)+r];
			gray = (gray > 0 ? gray : 0);
			gray = (gray < 0 ? 0 : gray)*scales / m;
			//fprintf(stdout, "%3ld ", table[c*(m+1)+r]);
			fprintf(stdout, "%c ", grayscale[gray]);
		}
		fprintf(stdout, "\n");
	}
	fprintf(stdout, "\n");
	free(table);
	hipFree(devtable);
#endif

	hipFree(devweftbuff);
	hipFree(devinframe);
	hipFree(devoutframe);

	result = outbound[n];

	return result;
}

__global__ void cu_init_row(long * row, const long n, const long m) {
	long thix = blockDim.x * blockIdx.x + threadIdx.x;

	for (int rep = 0; rep < n / m; ++rep) {
		if (rep*(m + 1) + thix < n + 1) {
			row[thix] = thix;
		}
	}
	if (thix < m) {
		row[n+2+thix] = thix+1;
	}
	__syncthreads();
}

// assuming the table array size (n+1) x (m+1)
__global__ void cu_dptable(long * weftbuff, const long * inframe, long * outframe, 
	const char t[], const long n, const char p[], const long m
#ifdef DEBUG_DPTABLE
	,long * table
#endif
) {
	long weft;
	long col, row, dix;
	long ins, del, repl, cellval;

	long thix = blockDim.x * blockIdx.x + threadIdx.x ;

	// get row = 0 or col = 0 values for each thread
	__syncthreads();

	// weft x thread
	for (weft = 0; weft < n + m + 1; ++weft) {
		if (thix < 2*(m + 1) ) {
			// dix = row - col.
			// thix = (dix + (m+1)) % (m+1)
			// col + row = weft whenever.
		}
		__syncthreads();
	}

	return;
}
