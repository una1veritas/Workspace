#include "hip/hip_runtime.h"
#include "cu_utils.h"

/*
#define NLZ32DEV(x) __clz(x)
#define NLZ64DEV(x) __clzll(x)
#define POPC32DEV(x) __popc(x)
#define POPC64DEV(x) __popcll(x)
*/

__device__ uint32 c2pow32dev(uint32 x) {
	return (x == 0) ? 0 : (1 << (32 - __clz(x - 1)));
}

__device__ uint32 clog32dev(uint32 x) {
	return (x == 0) ? 0 : (32 - __clz(x - 1));
}

__device__ uint32 bitsize32dev(int32 x) {
	x = x ^ (x >> 31);
	return 33 - __clz(x);
}

unsigned int nlz32_IEEEFP(unsigned int x)
{
	/* Hacker's Delight 2nd by H. S. Warren Jr., 5.3, p. 104 -- */
	double d = (double)x + 0.5;
	unsigned int *p = ((unsigned int*)&d) + 1;
	return 0x41e - (*p >> 20);  // 31 - ((*(p+1)>>20) - 0x3FF)
}

uint32 c2pow32(uint32 x) {
	return (x != 0) * (1 << (32 - NLZ32(x - 1)));

}

uint32 f2pow32(uint32 x) {
	return (x != 0) * (1 << (31 - NLZ32(x)));

}

uint32 flog32(uint32 x) {
	return 31 - NLZ32(x);

}

uint32 clog32(uint32 x) {
	return 32 - NLZ32(x - 1);
}

uint32 bitsize32(int32 x) {
	x = x ^ (x >> 31);
	return 33 - NLZ32(x);
}
