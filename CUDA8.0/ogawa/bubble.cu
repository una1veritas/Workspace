#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <time.h>

//#include <malloc.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include <helper_timer.h>

#define ARRAY_SIZE 16
//262144
#define BLOCK_SIZE 512

__global__ void bubSortEven(unsigned int *A, unsigned int *B);
__global__ void bubSortOdd(unsigned int *A, unsigned int *B);

int main(void) {

	if (((ARRAY_SIZE - 1) & ARRAY_SIZE) != 0) {
		printf("ARRAY_SIZE --- exponent of 2\n");
		return 0;
	}

	// device(0) : GTX 1080
	// device(1) : GTX 750Ti
	hipSetDevice(0);

	unsigned int  arraySize = sizeof(unsigned int) * ARRAY_SIZE;

	unsigned int *hA, *hB;
	hA = (unsigned int*)malloc(arraySize);
	hB = (unsigned int*)malloc(arraySize);

	unsigned int i;

	srand(time(NULL));

	for (i = 0; i < ARRAY_SIZE; i++) {
		hA[i] = rand() % 1024;

	}

	printf("Input:\n");
	for (i = 0; i < (ARRAY_SIZE < 100 ? ARRAY_SIZE : 100); i++)
		printf("%d, ", hA[i]);
	printf("\n\n");

	unsigned int *dA, *dB;
	hipMalloc((void**)&dA, arraySize);
	hipMalloc((void**)&dB, arraySize);

	hipMemcpy(dA, hA, arraySize, hipMemcpyHostToDevice);

	int blocks = ARRAY_SIZE + (2*BLOCK_SIZE - 1) / (2 * BLOCK_SIZE);
	int threads = BLOCK_SIZE;
	dim3 grid(blocks), block(threads);
	printf("grid dim = %d, block dim = %d\n", blocks, threads);

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	for (i = 0; i < 2; i++) {
		bubSortEven << < grid, block >> > (dA, dB);
		//hipDeviceSynchronize();
		bubSortOdd << < grid, block >> > (dB, dA);
		//hipDeviceSynchronize();
		hipError_t cuerr = hipGetLastError();
		if (cuerr != hipSuccess) {
			printf("cuda error: %s\n", hipGetErrorString(cuerr));
		}
	}

	hipMemcpy(hA, dA, arraySize, hipMemcpyDeviceToHost);
	hipMemcpy(hB, dB, arraySize, hipMemcpyDeviceToHost);

	sdkStopTimer(&timer);

	printf("Output:\n");
	for (i = 0; i < (ARRAY_SIZE < 100 ? ARRAY_SIZE : 100); i++) {
		//printf("[%d] = %u\n", i, hA[i]);
		printf("%d, ", hB[i]);
	}
	printf("\n");
	printf("[%d] = %u\n", ARRAY_SIZE - 1, hA[ARRAY_SIZE - 1]);

	printf("[TIME] =%f(ms)\n", sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);

	free(hA);
	free(hB);
	hipFree(dA);
	hipFree(dB);

	hipDeviceReset();

}

__global__ void bubSortEven(unsigned int *bef, unsigned int *aft) {

	unsigned int fro_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	unsigned int beh_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;

	if (bef[fro_idx] > bef[beh_idx]) {
		aft[fro_idx] = bef[beh_idx];
		aft[beh_idx] = bef[fro_idx];
	}
	else {
		aft[fro_idx] = bef[fro_idx];
		aft[beh_idx] = bef[beh_idx];
	}
	__syncthreads();
}

__global__ void bubSortOdd(unsigned int *bef, unsigned int *aft) {

	unsigned int fro_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;
	unsigned int beh_idx = fro_idx + 1;

	if (beh_idx != ARRAY_SIZE) {
		if (bef[fro_idx] > bef[beh_idx]) {
			aft[fro_idx] = bef[beh_idx];
			aft[beh_idx] = bef[fro_idx];
		}
		else {
			aft[fro_idx] = bef[fro_idx];
			aft[beh_idx] = bef[beh_idx];
		}
	}
	aft[0] = bef[0];
	aft[ARRAY_SIZE - 1] = bef[ARRAY_SIZE - 1];

	__syncthreads();
}
